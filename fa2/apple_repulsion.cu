
#include <hip/hip_runtime.h>
#include <stdio.h>

// Define a structure for nodes
struct Node {
    float x, y;    // Node coordinates
    float mass;    // Node mass
    float dx, dy;  // Node velocities
};

// CUDA kernel function for linear repulsion
__global__ void linear_repulsion_kernel(Node *nodes, int num_nodes, float coefficient) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < num_nodes && j < i) {
        float x_dist = nodes[i].x - nodes[j].x;
        float y_dist = nodes[i].y - nodes[j].y;
        float distance2 = x_dist * x_dist + y_dist * y_dist;

        if (distance2 > 0) {
            float factor = coefficient * nodes[i].mass * nodes[j].mass / distance2;
            nodes[i].dx += x_dist * factor;
            nodes[i].dy += y_dist * factor;
            nodes[j].dx -= x_dist * factor;
            nodes[j].dy -= y_dist * factor;
        }
    }
}

// Host function to apply repulsion on nodes using CUDA
void apply_repulsion2_cuda(Node *nodes, int num_nodes, float coefficient) {
    Node *d_nodes;
    hipMalloc((void **)&d_nodes, num_nodes * sizeof(Node));
    hipMemcpy(d_nodes, nodes, num_nodes * sizeof(Node), hipMemcpyHostToDevice);

    dim3 block_size(16, 16);  // Adjust the block size based on your requirements
    dim3 grid_size((num_nodes + block_size.x - 1) / block_size.x, (num_nodes + block_size.y - 1) / block_size.y);

    linear_repulsion_kernel<<<grid_size, block_size>>>(d_nodes, num_nodes, coefficient);

    hipMemcpy(nodes, d_nodes, num_nodes * sizeof(Node), hipMemcpyDeviceToHost);
    hipFree(d_nodes);
}

int main() {
    // Example usage
    const int num_nodes = 100;
    Node nodes[num_nodes];

    // Initialize nodes...

    float coefficient = 0.1;

    // Apply repulsion using CUDA
    apply_repulsion2_cuda(nodes, num_nodes, coefficient);

    // Rest of your code...

    return 0;
}